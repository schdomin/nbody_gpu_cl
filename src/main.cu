#include "hip/hip_runtime.h"
#include "CCubicDomain.cuh"           //ds domain structure
#include "Timer.cuh"                  //ds time measurement
#include <iostream>                   //ds cout
#include <hip/hip_runtime.h>                     //ds needed for eclipse indexer only (not for compilation)
#include <hip/hip_runtime.h>             //ds needed for eclipse indexer only (not for compilation)
#include <> //ds needed for eclipse indexer only (not for compilation)



//ds cuda kernels - optimized for one block
__global__ void computeAccelerationsLennardJones( const NBody::CParticle* p_vecParticles,
                                                  const std::pair< unsigned int, unsigned int >* p_arrCellIndexRange,
                                                  const double p_dMinimumDistance,
                                                  const double p_dPotentialDepth,
                                                  const unsigned int p_uMaximumCellIndex,
                                                  const unsigned int p_uMaximumNeighborCellIndexRange,
                                                  double* p_arrNewAccelerations )
{
    //ds get the current cell index
    const unsigned int uCurrentCellIndex( threadIdx.x );

    //ds handle the current cell particles - this only works because the data is ordered also in cell order (indexing is shifted by 1, [0,0] means there is no particle in the cell)
    for( unsigned int v = p_arrCellIndexRange[uCurrentCellIndex].first; v < p_arrCellIndexRange[uCurrentCellIndex].second; ++v )
    {
        //ds get the particle index
        const unsigned int uCurrentParticleIndex1D( ( v-1 ) );

        //ds for each particle in this cell
        for( int i = static_cast< int >( uCurrentCellIndex-p_uMaximumNeighborCellIndexRange ); i < static_cast< int >( uCurrentCellIndex+p_uMaximumNeighborCellIndexRange+1 ); ++i )
        {
            //ds force to sum up
            double vecTotalForce[3];

            //ds initialize the vector
            vecTotalForce[0] = 0.0;
            vecTotalForce[1] = 0.0;
            vecTotalForce[2] = 0.0;

            //ds internal "real" index
            unsigned int uNeighborCellIndex( 0 );

            //ds check for periodic boundary conditions
            if( i < 0 )
            {
                //ds shift up by maximum cell index
                uNeighborCellIndex += p_uMaximumCellIndex;
            }
            else if( i > static_cast< int >( p_uMaximumCellIndex ) )
            {
                //ds shift down by maximum cell index
                uNeighborCellIndex -= p_uMaximumCellIndex;
            }
            else
            {
                //ds regular case (i must be positive)
                uNeighborCellIndex = i;
            }

            //ds interact with all neighbor particles
            for( unsigned int w = p_arrCellIndexRange[uNeighborCellIndex].first; w < p_arrCellIndexRange[uNeighborCellIndex].second; ++w )
            {
                //ds get the radial vector between the particles
                double vecRadius[3];

                //ds calculate the distance: domain + particle2 - particle1
                vecRadius[0] = p_vecParticles[w-1].m_cPosition[0] - p_vecParticles[uCurrentParticleIndex1D].m_cPosition[0];
                vecRadius[1] = p_vecParticles[w-1].m_cPosition[1] - p_vecParticles[uCurrentParticleIndex1D].m_cPosition[1];
                vecRadius[2] = p_vecParticles[w-1].m_cPosition[2] - p_vecParticles[uCurrentParticleIndex1D].m_cPosition[2];

                //ds get the absolute distance
                const double dDistanceAbsolute( sqrt( pow( vecRadius[0], 2 ) + pow( vecRadius[1], 2 ) + pow( vecRadius[2], 2 ) ) );

                //ds calculate the lennard jones force prefix
                const double dLJFPrefix( -24*p_dPotentialDepth*( 2*pow( p_dMinimumDistance/dDistanceAbsolute, 12 ) - pow( p_dMinimumDistance/dDistanceAbsolute, 6  ) )
                                                              *1/pow( dDistanceAbsolute, 2 ) );

                //ds add the information to the force including the radial component
                vecTotalForce[0] += dLJFPrefix*vecRadius[0];
                vecTotalForce[1] += dLJFPrefix*vecRadius[1];
                vecTotalForce[2] += dLJFPrefix*vecRadius[2];
            }

            //ds get particle mass
            const double dParticleMass( p_vecParticles[uCurrentParticleIndex1D].m_dMass );

            //ds if we got the total force calculate the resulting acceleration and save it to our array
            p_arrNewAccelerations[3*uCurrentParticleIndex1D+0] = vecTotalForce[0]/dParticleMass;
            p_arrNewAccelerations[3*uCurrentParticleIndex1D+1] = vecTotalForce[1]/dParticleMass;
            p_arrNewAccelerations[3*uCurrentParticleIndex1D+2] = vecTotalForce[2]/dParticleMass;
        }
    }
}

__global__ void updateParticlesVelocityVerlet( NBody::CParticle* p_vecParticles,
                                               const double* p_arrNewAccelerations,
                                               const double p_dLowerBoundary,
                                               const double p_dUpperBoundary,
                                               const double p_dTimeStepSize )
{
    //ds particle index
    const unsigned int uIndex1D( threadIdx.x );

    //ds calculate domain size
    const double dDomainSize( fabs( p_dLowerBoundary ) + fabs( p_dUpperBoundary ) );

    //ds get properties
    double* vecPosition              = p_vecParticles[uIndex1D].m_cPosition;
    double* vecVelocity              = p_vecParticles[uIndex1D].m_cVelocity;
    double* vecAcceleration          = p_vecParticles[uIndex1D].m_cAcceleration;
    const double* vecNewAcceleration = &p_arrNewAccelerations[3*uIndex1D];

    //ds velocity-verlet for position
    vecPosition[0] = vecPosition[0] + p_dTimeStepSize*vecVelocity[0] + 1.0/2*pow( p_dTimeStepSize, 2 )*vecAcceleration[0];
    vecPosition[1] = vecPosition[1] + p_dTimeStepSize*vecVelocity[1] + 1.0/2*pow( p_dTimeStepSize, 2 )*vecAcceleration[1];
    vecPosition[2] = vecPosition[2] + p_dTimeStepSize*vecVelocity[2] + 1.0/2*pow( p_dTimeStepSize, 2 )*vecAcceleration[2];

    //ds produce periodic boundary shifting - check each element: x,y,z
    for( unsigned int v = 0; v < 3; ++v )
    {
        //ds check if we are below the boundary
        while( p_dLowerBoundary > vecPosition[v] )
        {
            //ds map the particle to the other boundary by shifting it up to the boundary
            vecPosition[v] += dDomainSize;
        }

        //ds check if we are above the boundary
        while( p_dUpperBoundary < vecPosition[v] )
        {
            //ds map the particle to the other boundary by shifting it back to the boundary
            vecPosition[v] -= dDomainSize;
        }
    }

    //ds velocity-verlet for velocity
    vecVelocity[0] = vecVelocity[0] + ( p_dTimeStepSize/2 )*( vecNewAcceleration[0] + vecAcceleration[0] );
    vecVelocity[1] = vecVelocity[1] + ( p_dTimeStepSize/2 )*( vecNewAcceleration[1] + vecAcceleration[1] );
    vecVelocity[2] = vecVelocity[2] + ( p_dTimeStepSize/2 )*( vecNewAcceleration[2] + vecAcceleration[2] );

    //ds update the old accelerations
    vecAcceleration[0] = vecNewAcceleration[0];
    vecAcceleration[1] = vecNewAcceleration[1];
    vecAcceleration[2] = vecNewAcceleration[2];
}

int main( int argc, char** argv )
{
    //ds start timing
    Timer tmTimer; tmTimer.start( );

    //ds domain configuration
    const std::pair< double, double > pairBoundaries( -1.0, 1.0 );
    const double dDomainWidth( fabs( pairBoundaries.first ) + fabs( pairBoundaries.second ) );
    const unsigned int uNumberOfParticles( 100 );

    //ds current simulation configuration
    const double dTimeStepSize( 0.0001 );
    const unsigned int uNumberOfTimeSteps( 5000 );
    const double dMinimumDistance( pow( 1.0/uNumberOfParticles, 1.0/3 ) );
    const double dPotentialDepth( 1.0 );

    //ds target kinetic energy
    const double dTargetKineticEnergy( 1000.0 );

    //ds cell list information
    const unsigned int uNumberOfCells1D( floor( dDomainWidth/( 2.5*dMinimumDistance ) ) );
    const unsigned int uMaximumCellIndex( uNumberOfCells1D + pow( uNumberOfCells1D, 2 ) + pow( uNumberOfCells1D, 3 ) + 1 );

    std::cout << "------- GPU SETUP -----------------------------------------------------------" << std::endl;
    std::cout << "  Number of particles: " << uNumberOfParticles << std::endl;
    std::cout << "        Boundary (3D): [" << pairBoundaries.first << ", " << pairBoundaries.second << "]" << std::endl;
    std::cout << "         Domain Width: " << dDomainWidth << std::endl;
    std::cout << "     Minimum distance: " << dMinimumDistance << std::endl;
    std::cout << "      Cutoff distance: " << 2.5*dMinimumDistance << std::endl;
    std::cout << "      Potential depth: " << dPotentialDepth << std::endl;
    std::cout << "Target kinetic energy: " << dTargetKineticEnergy << std::endl;
    std::cout << " Number of time steps: " << uNumberOfTimeSteps << std::endl;
    std::cout << "       Time step size: " << dTimeStepSize << std::endl;
    std::cout << "------- CELL LISTS ----------------------------------------------------------" << std::endl;
    std::cout << " Number of cells 1D M: " << uNumberOfCells1D << std::endl;
    std::cout << "   Maximum cell index: " << uMaximumCellIndex << std::endl;
    std::cout << "-----------------------------------------------------------------------------" << std::endl;

    //ds allocate a domain to work with specifying number of particles and timing
    NBody::CCubicDomain cDomain( pairBoundaries, uNumberOfParticles, dMinimumDistance, uNumberOfCells1D, uMaximumCellIndex );

    //ds create particles uniformly from a normal distribution
    cDomain.createParticlesUniformFromNormalDistribution( dTargetKineticEnergy );

    //ds support structure for host and device
    std::pair< unsigned int, unsigned int > *h_arrCellIndexRange = cDomain.getCellIndexRange( );
    std::pair< unsigned int, unsigned int > *d_arrCellIndexRange = 0;

    //ds accelerations buffer on the GPU
    double* d_arrNewAccelerations( 0 ); //Nx3

    //ds allocate memory on device
    hipMalloc( (void **)&d_arrCellIndexRange, uMaximumCellIndex*sizeof( std::pair< unsigned int, unsigned int > ) );
    hipMalloc( (void **)&d_arrNewAccelerations, uNumberOfParticles*3*sizeof( double ) ) ;

    //ds get particles for the device
    thrust::device_vector< NBody::CParticle > d_vecParticles( cDomain.getParticles( ) );

    //ds get a raw pointer for kernel usage
    NBody::CParticle *vecParticles( thrust::raw_pointer_cast( &d_vecParticles[0] ) );

    //ds get cell list specific parameters
    const unsigned int uMaximumNeighborCellIndexRange( cDomain.getMaximumNeighborCellIndexRange( ) );

    //ds information
    std::cout << "               Status:  0% done - current step: 0";

    //ds start simulation
    for( unsigned int uCurrentTimeStep = 1; uCurrentTimeStep < uNumberOfTimeSteps+1; ++uCurrentTimeStep )
    {
        //ds calculate percentage done
        const double dPercentageDone( 100.0*uCurrentTimeStep/uNumberOfTimeSteps );

        //ds get a formatted string -> 100% -> 3 digits
        char chBuffer[4];

        //ds fill the buffer
        std::snprintf( chBuffer, 4, "%3.0f", dPercentageDone );

        //ds print info
        std::cout << '\xd';
        std::cout << "               Status: " << chBuffer << "% done - current step: " << uCurrentTimeStep;

        //ds copy support structure memory to gpu
        hipMemcpy( d_arrCellIndexRange, h_arrCellIndexRange, uMaximumCellIndex*sizeof( std::pair< unsigned int, unsigned int > ), hipMemcpyHostToDevice );

        //ds compute accelerations for all cells - launch as many threads as we have cells
        computeAccelerationsLennardJones<<< 1, uMaximumCellIndex >>>( vecParticles,
                                                                      d_arrCellIndexRange,
                                                                      dMinimumDistance,
                                                                      dPotentialDepth,
                                                                      uMaximumCellIndex,
                                                                      uMaximumNeighborCellIndexRange,
                                                                      d_arrNewAccelerations );
        //ds update particle properties
        updateParticlesVelocityVerlet<<< 1, uNumberOfParticles >>>( vecParticles,
                                                                    d_arrNewAccelerations,
                                                                    pairBoundaries.first,
                                                                    pairBoundaries.second,
                                                                    dTimeStepSize );

        //ds copy particles back to the domain - this call also updates the cell lists and changes the support structure
        cDomain.setParticles( d_vecParticles );

        //ds record situation (we will write the stream to the file in one operation afterwards )
        cDomain.saveParticlesToStream( );
        cDomain.saveIntegralsToStream( dMinimumDistance, dPotentialDepth );
    }

    //ds deallocate memory
    hipFree( d_arrCellIndexRange );
    hipFree( d_arrNewAccelerations );

    //ds save the streams to a file
    cDomain.writeParticlesToFile( "bin/simulation.txt", uNumberOfTimeSteps );
    cDomain.writeIntegralsToFile( "bin/integrals.txt", uNumberOfTimeSteps, dTimeStepSize );

    //ds stop timing
    const double dDurationSeconds( tmTimer.stop( ) );

    //ds cause an output ostream
    std::cout << std::endl;
    std::cout << "     Computation time: " << dDurationSeconds << std::endl;
    std::cout << "-----------------------------------------------------------------------------" << std::endl;

    return 0;
}
